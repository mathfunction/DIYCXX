#include "hip/hip_runtime.h"
/*=================================================================



 __host__   : CPU 調用的 普通函數
 __global__ : CPU 執行到 GPU 得分派函式 	// 實作無法使用常見函式庫
 __device__	: GPU 調用的函式 			// 實作無法使用常見函式庫

 同個 Block 裡面的 threads shard memory !! 
 	blockIdx.x = [0,blocksPerGrid)     block may be coordinate (x,y,z)
 	blockDim.x = threadsPerBlock	   
 	threadIdx.x = [0,threadsPerBlock)  threads may be coordinate (x,y,z)

	<<< blocksPerGrid , threadsPerBlock >>>

====================================================================*/




#include "cxxuseful.hpp"
#include "npp.h"




using namespace cxxuseful;


const int size = 10;
const size_t bytes = size*sizeof(float);

// 平行化實作
__global__ void vecadd(const float *v1,const float *v2,float *v3){
	// 計算 thread id 
	int id = blockIdx.x*blockDim.x+threadIdx.x;
	// 只作用 size 以下的運算
	if(id < size){
		v3[id] = v1[id] + v2[id];
	}//endif
}




// 如何平行化這個程式
__host__ void vecadd_driver(const float *v1,const float *v2,float *v3){
  int threadsPerBlock = size;
  int blocksPerGrid = 1;
  //int threadsPerBlock = 1;
  //int blocksPerGrid = size;
  vecadd<<<blocksPerGrid,threadsPerBlock>>>(v1,v2,v3);
  
  printYellow("Hello vector addtiion !!! ",true);
}



int main(){
	
	
	// CPU 動態配置向量 
	float* c1 = (float*)malloc(bytes);
  	float* c2 = (float*)malloc(bytes);
  	float* c3 = (float*)malloc(bytes);


  	// GPU 動態配置向量
  	float*g1,*g2,*g3;
  	hipMalloc(&g1,bytes);
  	hipMalloc(&g2,bytes);
  	hipMalloc(&g3,bytes);

  	// 賦值
  	for(int i=0;i<size;i++){
  		c1[i] = float(i);
   		c2[i] = float(i);
  	}//endfor
  	// 把 CPU 向量的值複製到 GPU  , 要放置 bytes 進去 
  	hipMemcpy(g1,c1,bytes,hipMemcpyHostToDevice);
  	hipMemcpy(g2,c2,bytes,hipMemcpyHostToDevice);
  	//====================================================================
  	vecadd_driver(g1,g2,g3);
  	// 回傳答案 
  	hipMemcpy(c3,g3,bytes,hipMemcpyDeviceToHost);
  	for(int i=0;i<size;i++){
  		std::cout << "==================================================" << std::endl;
  		std::cout << "c1[" << i <<  "] = " << c1[i]  << std::endl;
  		std::cout << "c2[" << i <<  "] = " << c2[i]  << std::endl;
  		std::cout << "c3[" << i <<  "] = " << c3[i]  << std::endl;
  	}//endfor

  	//===================================================================
  	// 釋放記憶體 GPU , CPU 
  	free(c1);
  	free(c2);
  	free(c3);
  	hipFree(g1);
  	hipFree(g2);
  	hipFree(g3);


	return 0;
}//end_main