#include "hip/hip_runtime.h"


#include "cxxuseful.hpp"


using namespace cxxuseful;


extern "C" {
	#include <stdio.h>
	__global__ void helloworld(){
		 printf("Hello world!\n");
	}
}


void cudaInit(void){
	helloworld<<<1,1>>>();
}



int main(){

	cudaInit();
	printGreen("______This is Green________",true);


	return 0;
}